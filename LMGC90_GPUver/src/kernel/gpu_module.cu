
#include <hip/hip_runtime.h>

__global__ void matmul_kernel(double *A, double *B, double *C, int N) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N && j < N) {
    double sum = 0.0;
    for (int k = 0; k < N; k++) {
      sum += A[i * N + k] * B[k * N + j];
    }
    C[i * N + j] = sum;
  }
}

extern "C" void matmul_cuda(double *A, double *B, double *C, int N) {
  // Allocate device memory
  double *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, N*N*sizeof(double));
  hipMalloc((void**)&d_B, N*N*sizeof(double));
  hipMalloc((void**)&d_C, N*N*sizeof(double));

  // Copy data to device
  hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice);

  // Define block and grid sizes
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                 (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Launch kernel
  matmul_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result back to host
  hipMemcpy(C, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
